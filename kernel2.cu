#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2022 Bogdan Simion
 * -------------
 */

#include "kernels.h"
#include <cstdint>

GpuTimes run_kernel2(const int8_t* filter, int32_t dimension, const int32_t* input,
                     int32_t* output, int32_t width, int32_t height)
{
    GpuTimer timer = GpuTimer(2);
    timer.start();

    int32_t *d_input, *d_output;
    int8_t* d_filter;
    int32_t filter_size = dimension * dimension * sizeof(int8_t);
    int32_t image_size_bytes = width * height * sizeof(int32_t);

    // Launching the kernel with one thread per pixel in row-major order
    int32_t threads_per_block = THREADS_PER_BLOCK;
    int32_t total_blocks = (width * height + threads_per_block - 1) / threads_per_block;

    int32_t* d_min_array = nullptr;
    int32_t* d_max_array = nullptr;
    int32_t* h_min_array = new int32_t[total_blocks];
    int32_t* h_max_array = new int32_t[total_blocks];

    allocate_device_memory(d_input, d_output, image_size_bytes, d_filter, filter_size, d_min_array, d_max_array,
                           total_blocks);

    transfer_to_device(input, d_input, image_size_bytes, filter, d_filter, filter_size, timer);

    timer.start_compute();
    kernel2<<<total_blocks, threads_per_block>>>(d_filter, dimension, d_input, d_output, d_min_array, d_max_array,
                                                 width, height);

    int32_t h_min, h_max;
    reduce_min_max(h_min_array, d_min_array, h_min, h_max_array, d_max_array, h_max, total_blocks);

    normalize2<<<total_blocks, threads_per_block>>>(d_output, width, height, h_min, h_max);
    timer.stop_compute();

    transfer_to_host(output, timer, d_output, image_size_bytes);

    free_memory(d_input, d_output, d_filter, d_min_array, d_max_array, h_min_array, h_max_array);

    timer.stop();

    return timer.get_times();
}

__global__ void kernel2(const int8_t* filter, int32_t dimension, const int32_t* input, int32_t* output,
                        int32_t* d_min, int32_t* d_max, int width, int height)
{
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;

    __shared__ int min_cache[THREADS_PER_BLOCK];
    __shared__ int max_cache[THREADS_PER_BLOCK];

    int temp_min = INT_MAX;
    int temp_max = INT_MIN;

    // Each thread processes one pixel in row-major order
    while (thread_id < width * height)
    {
        int row = thread_id / width;
        int col = thread_id % width;

        int sum = 0;
        for (int i = 0; i < dimension; i++)
        {
            for (int j = 0; j < dimension; j++)
            {
                int x = col + j - dimension / 2;
                int y = row + i - dimension / 2;
                int pixel = 0;
                if (x >= 0 && x < width && y >= 0 && y < height)
                {
                    pixel = input[y * width + x];
                }
                sum += filter[i * dimension + j] * pixel;
            }
        }
        output[row * width + col] = sum;
        temp_min = min(temp_min, sum);
        temp_max = max(temp_max, sum);
        thread_id += total_threads;
    }
    min_cache[threadIdx.x] = temp_min;
    max_cache[threadIdx.x] = temp_max;
    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i >>= 1)
    {
        if (threadIdx.x < i)
        {
            min_cache[threadIdx.x] = min(min_cache[threadIdx.x], min_cache[threadIdx.x + i]);
            max_cache[threadIdx.x] = max(max_cache[threadIdx.x], max_cache[threadIdx.x + i]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        d_min[blockIdx.x] = min_cache[0];
        d_max[blockIdx.x] = max_cache[0];
    }
}

__global__ void normalize2(int32_t* image, int32_t width, int32_t height,
                           int32_t smallest, int32_t biggest)
{
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;

    while (thread_id < width * height)
    {
        int row = thread_id / width;
        int col = thread_id % width;

        int index = row * width + col;
        if (biggest != smallest)
        {
            float normalized = 255.0f * (image[index] - smallest) / (biggest - smallest);
            image[index] = static_cast<int32_t>(normalized);
        }
        // Make sure between 0 and 255
        image[index] = max(0, min(255, image[index]));
        thread_id += total_threads;
    }
}
